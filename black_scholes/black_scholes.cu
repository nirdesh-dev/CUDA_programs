#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define N 1000000  // 1 million options

// Normal cumulative distribution function (CNDF)
__device__ double normal_cdf(double x) {
    return 0.5 * (1.0 + erf(x / sqrt(2.0)));
}

// Black-Scholes kernel running on the GPU
__global__ void black_scholes_kernel(double *d_S, double *d_K, double *d_T, 
                                     double r, double sigma, 
                                     double *d_call, double *d_put) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < N) {
        double S = d_S[idx];
        double K = d_K[idx];
        double T = d_T[idx];

        double d1 = (log(S / K) + (r + 0.5 * sigma * sigma) * T) / (sigma * sqrt(T));
        double d2 = d1 - sigma * sqrt(T);

        d_call[idx] = S * normal_cdf(d1) - K * exp(-r * T) * normal_cdf(d2);
        d_put[idx] = K * exp(-r * T) * normal_cdf(-d2) - S * normal_cdf(-d1);
    }
}

int main() {
    // Allocate memory on host (CPU)
    double *h_S = (double*)malloc(N * sizeof(double));
    double *h_K = (double*)malloc(N * sizeof(double));
    double *h_T = (double*)malloc(N * sizeof(double));
    double *h_call = (double*)malloc(N * sizeof(double));
    double *h_put = (double*)malloc(N * sizeof(double));

    double r = 0.01 + ((double)rand() / RAND_MAX) * 0.1;  // Random risk-free rate (1% to 11%)
    double sigma = 0.1 + ((double)rand() / RAND_MAX) * 0.5;  // Random volatility (10% to 60%)

    // Initialize input data with randomized values
    for (int i = 0; i < N; i++) {
        h_S[i] = 50.0 + ((double)rand() / RAND_MAX) * 200.0;  // Stock price (50 to 250)
        h_K[i] = 50.0 + ((double)rand() / RAND_MAX) * 200.0;  // Strike price (50 to 250)
        h_T[i] = 0.1 + ((double)rand() / RAND_MAX) * 5.0;  // Time to expiry (0.1 to 5 years)
    }

    // Allocate memory on GPU
    double *d_S, *d_K, *d_T, *d_call, *d_put;
    hipMalloc((void**)&d_S, N * sizeof(double));
    hipMalloc((void**)&d_K, N * sizeof(double));
    hipMalloc((void**)&d_T, N * sizeof(double));
    hipMalloc((void**)&d_call, N * sizeof(double));
    hipMalloc((void**)&d_put, N * sizeof(double));

    // Copy data to GPU
    hipMemcpy(d_S, h_S, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_K, h_K, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_T, h_T, N * sizeof(double), hipMemcpyHostToDevice);

    // Launch kernel with optimized grid size
    int threads_per_block = 1024;
    int blocks_per_grid = (N + threads_per_block - 1) / threads_per_block;
    black_scholes_kernel<<<blocks_per_grid, threads_per_block>>>(d_S, d_K, d_T, r, sigma, d_call, d_put);

    // Copy results back to CPU
    hipMemcpy(h_call, d_call, N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_put, d_put, N * sizeof(double), hipMemcpyDeviceToHost);

    // Write results to file
    FILE *file = fopen("output.txt", "w");
    if (file == NULL) {
        printf("Error opening file!\n");
        return 1;
    }

    fprintf(file, "Stock Price | Strike Price | Time to Expiry | Call Price | Put Price\n");
    fprintf(file, "-------------------------------------------------------------\n");
    for (int i = 0; i < N; i++) {
        fprintf(file, "%11.2f | %12.2f | %14.2f | %10.5f | %9.5f\n", 
                h_S[i], h_K[i], h_T[i], h_call[i], h_put[i]);
    }
    fclose(file);

    printf("Results written to output.txt\n");

    // Free GPU and CPU memory
    hipFree(d_S);
    hipFree(d_K);
    hipFree(d_T);
    hipFree(d_call);
    hipFree(d_put);
    free(h_S);
    free(h_K);
    free(h_T);
    free(h_call);
    free(h_put);

    return 0;
}
